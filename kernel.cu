#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

#include <iostream>
#include <thread>
#include <chrono>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define NUM_TEAMS 400
#define RUNNER_COUNT 4
#define DISTANCE 400


/*
* 1 ile 5 arasında rastgele bir float sayı üretir.
* GPU'dan çalıştırmak için __device__ kullanır ve GPU'dan da çağrılacaktır.
*/
__device__ float generateRandomNumber(hiprandState* state) {

    return (hiprand_uniform(state) * 4 + 1);
}
/*
* Bayrak yarışı için algoritma.
* Ayrıca:
    - Her takım için kaç saniyenin geçtiğini hesaplar,
    - Herhangi bir koşucunun bitiş sırasını
    - Anlık olarak o anki takımın hangi koşuşunun koştuğunu  (1. 2. 3. veya 4.)
    - Ve koşucuların katettiği mesafeyi de hesaplar

  finishedOrder kaç takımın yarışı bitirdiğini hesaplamak içindir.
*
* GPU'dan çalıştırmak için __global__ kullanır ve CPU'dan çağrılır.
*/
__global__ void race(float* distances, int* currentRunner, int* placements, int* finishedOrder, int* seconds) {

    int index = threadIdx.x;

    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandState state;
    hiprand_init(0, threadId, 0, &state);

    float speed = generateRandomNumber(&state);


    if (distances[index] < DISTANCE) {

        if (distances[index] + speed >= DISTANCE) {
            distances[index] = DISTANCE;
            placements[index] = finishedOrder[0] + 1;
            finishedOrder[0]++;
        }
        else {
            distances[index] += speed;
        }


        // Bayrak degisimi icin yazılmıs fonksiyon
        int checkpoint = (DISTANCE / RUNNER_COUNT) * currentRunner[index];

        if (distances[index] >= checkpoint) {

            if (currentRunner[index] < RUNNER_COUNT)
                currentRunner[index]++;
        }
        seconds[index]++;

    }

}


/*
* Takımların kat ettiği mesafelerden yarışın bitip bitmediğini kontrol eder.
*/
bool isRaceFinished(float* distances)
{
    for (int i = 0; i < NUM_TEAMS; i++) {
        if (distances[i] < DISTANCE) {
            return false;
        }
    }
    return true;
}


/*
* Şu anda yarışan her takımın mesafelerini yazdırır.
* printAll false ise sadece "takımCount" adet kosucuyu, "takımKosucuları" parametresinden çekerek yazdırır.
* aksi halde tüm kosucuları yazdırır.
*/
void printDistances(float* distances, int* currentRunner, int takimCount, int* takimSiralari, int* placements, int* seconds, bool printAll) {
    int takimIndex = 0;
    int minutes;
    int remainingSeconds;
    for (int i = 0; i < NUM_TEAMS; i++) {
        if (printAll)
        {
            if (placements[i] != 0) {
                if (seconds[i] > 60) {
                    minutes = seconds[i] / 60;
                    remainingSeconds = seconds[i] % 60;
                    std::cout << "Team " << i + 1 << ": Runner " << currentRunner[i] << " - " << " Finished at: " << placements[i] << "th place. " << " with " << minutes << " minutes " << remainingSeconds << ".seconds. \n" << std::endl;
                }
                else {
                    std::cout << "Team " << i + 1 << ": Runner " << currentRunner[i] << " - " << " Finished at: " << placements[i] << "th place. " << " with " << seconds[i] << " seconds.\n" << std::endl;
                }
            }
        }
        else
        {
            if (takimIndex < takimCount) {
                if (i + 1 == takimSiralari[takimIndex]) {
                    if (placements[i] != 0) {
                        if (seconds[i] > 60) {
                            minutes = seconds[i] / 60;
                            remainingSeconds = seconds[i] % 60;
                            std::cout << "Team " << i + 1 << ": Runner " << currentRunner[i] << " - " << " Finished at: " << placements[i] << "th place. " << " with " << minutes << " minutes " << remainingSeconds << ".seconds. \n";
                        }
                        else {
                            std::cout << "Team " << i + 1 << ": Runner " << currentRunner[i] << " - " << " Finished at: " << placements[i] << "th place. " << " with " << seconds[i] << " seconds. \n " << std::endl;
                        }
                    }
                    else {
                        std::cout << "Team " << i + 1 << ": Runner " << currentRunner[i] << " - " << distances[i] << " m's of distance traveled with speed of: " << (distances[i] / seconds[i]) << "m/s" << " with " << seconds[i] << ".seconds. \n" << std::endl;
                    }
                    takimIndex++;
                }
            }
        }
    }
}

/*
* Yarışı bitiren her takımın sıralamasını yazdırır.
* Parametre olarak geçirilen "takımCount" kadar koşucuyu "takımSıraları"'ndan çekerek yazdırır.
* Ayrıca yarışı bitirmeleri için kaç saniye yarıştıklarını de yazdırır.
*/
void printTeamsPlacements(int takimCount, int* takimSiralari, int* placements, int* seconds) {

    int takimIndex = 0;
    for (int i = 0; i < NUM_TEAMS; i++) {

        if (takimIndex < takimCount) {

            if (i + 1 == takimSiralari[takimIndex]) {

                if (seconds[i] != 0) {
                    if (seconds[i] > 60) {
                        int minutes = seconds[i] / 60;
                        int remainingSeconds = seconds[i] % 60;
                        std::cout << "Team " << i + 1 << " -- Finished at: " << placements[i] << "th place. With " << minutes << " minutes " << remainingSeconds << ".seconds.\n";
                    }
                    else {
                        std::cout << "Team " << i + 1 << " -- Finished at: " << placements[i] << "th place. With " << seconds[i] << " seconds.\n" << std::endl;

                    }
                }
                takimIndex++;
            }
        }
    }
}

int main() {


    float* distances;
    int* currentRunner;
    int* placements;
    int* seconds;
    int* finishedOrder;

    hipMallocManaged(&finishedOrder, sizeof(int));
    hipMallocManaged(&distances, NUM_TEAMS * sizeof(float));
    hipMallocManaged(&currentRunner, NUM_TEAMS * sizeof(int));
    hipMallocManaged(&seconds, NUM_TEAMS * sizeof(int));
    hipMallocManaged(&placements, NUM_TEAMS * sizeof(int));


    finishedOrder[0] = 0;


    std::cout << "Kac takimi takip etmek istiyorsunuz: ";
    int takimCount;

    std::cin >> takimCount;

    int* takimlar;
    hipMallocManaged(&takimlar, NUM_TEAMS * sizeof(int));

    std::cout << "Takip etmek istediğiniz takımların no'sunu giriniz: ";
    for (int i = 0; i < takimCount; i++) {
        std::cin >> takimlar[i];
        while (takimlar[i] < 0 || takimlar[i] > 400) {
            std::cout << "1 ila 400 arası (1 ile 400 dahil) bir takım numarası giriniz: ";
            std::cin >> takimlar[i];
        }
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    for (int i = 0; i < NUM_TEAMS; i++) {
        distances[i] = 0.f;
        currentRunner[i] = 1;
        placements[i] = 0;
        seconds[i] = 0;
    }

    while (!isRaceFinished(distances)) {
        race << <1, NUM_TEAMS >> > (distances, currentRunner, placements, finishedOrder, seconds);
        hipDeviceSynchronize();
        printDistances(distances, currentRunner, takimCount, takimlar, placements, seconds, false);
        std::this_thread::sleep_for(std::chrono::seconds(1));
        //system("CLS");
    }


    printDistances(distances, currentRunner, takimCount, takimlar, placements, seconds, true);

    std::cout << "\n\nYour teams placements: " << std::endl << std::endl;

    printTeamsPlacements(takimCount, takimlar, placements, seconds);


    hipFree(distances);
    hipFree(currentRunner);

    return 0;
}